#include "hip/hip_runtime.h"
/* @BEGIN_DOC_FILE!
File: EELSurrogate.cu
Purpose: This file contains the functions to implement a Kriging Surrogate model from an input data file.
Author: Joseph Haun
 @END_DOC_FILE!*/

//Surrogate Header File
#include "EELSurrogate.cuh"

//#include <hip/hip_complex.h> //Unable to find proper documentation for hipComplex functionality. As such, some of the documentation relating to complex values is meaningless. 

//Begin Function Implementations
int colMajIndex(int x, int y, int dimension) {
    return x + y * dimension; #Dimension is columns
}

int rowMajIndex(int x, int y, int dimension) {
    return x * dimension + y; #Dimension is rows
}

double* readInputFile(std::string fileName) {
    //Assume no header lines
    return readInputFile(fileName, 0);
}

double* readInputFile(std::string fileName, int headerLines) {

    //Create filestream object and open target file
    std::fstream fileObject;
    fileObject.open(fileName, std::fstream::in);

    //Declare vector for file data
    std::vector<double> tempValues;

    //Declare vector and strings for parsing
    std::vector<std::string> row;
    std::string line, word, temp;

    //Declare counters
    int curRowIndex = 0;
    int curColIndex = 0;
    int numRows = 0;
    int numCols = 0;
    int headersSkipped = 0;

    while (fileObject >> line) {
        //Clear the vector of all values
        row.clear();

        //Break words apart
        if (headersSkipped >= headerLines) {
            std::stringstream inStream(line);
            
            while (getline(inStream, word, ',')) {
	    	//Attempt to convert the word to a double.
                try {
                    tempValues.push_back(std::stod(word.c_str()));
                }
                catch (const std::exception& e) {
                    tempValues.push_back(0.0);
                    printf("\nException occurred while attempting to convert '%s' to a double value. \nThe resulting output matrix has (%d, %d) set to zero.\n", word, curRowIndex, curColIndex);
                }
                
                curColIndex++;
                
            }

	    if(curColIndex > numCols){
	        numCols = curColIndex;
	    }	    
	    
            //Reset column index and increment row count
            numRows++;
            curColIndex = 0;
        }
        else {
            headersSkipped++;
        }
        
    }

    fileObject.close();

    //Allocate memory on heap for returned array and set all indices to 0.0
    double* fileValues = (double*)malloc(tempValues.size() * sizeof(double));
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numCols; j++) {
            //printf("(%d,%d): %d -> %d\n", i, j, rowMajIndex(i,j,dimension), colMajIndex(i, j, fileLines));
            fileValues[colMajIndex(i, j, numRows)] = tempValues.at(rowMajIndex(i,j,numCols));
        }
    }

    return fileValues;
}

int* generateSample(int numRows, float size, bool random){

    //Generate the samples number for the array
    int numSamples = 0;
    //Enforce percentage limits
    if(size > 1.0 || size < 0.0){
       numSamples = numRows;
    }
    else{
       numSamples = numRows*size;
    }   

    //Allocate output memory
    int* samples (int*)malloc( numSamples* sizeof(int));
    
    //Initialize random seed
    std::srand(time(nullptr));

    bool duplicate = false;
    int checkCounter;
    for(int i = 0; i < numSamples; i++){
 	samples[i] = std::rand()%numRows;
	checkCounter = 0;
    	while(i > 0 && duplicate == false && checkCounter <= i){
	    //If the newly generated sample is already in the array
	    if(samples[checkCounter] == samples[i]){
	        //Generate new random value and reset counter
	        samples[i] = std::rand()%numRows;
		checkCounter = 0;
	    }
	    //Otherwise, increment counter
	    checkCounter++;
	}
    }
    return samples;
}

void parseRawData(double* outputParameters, double* outputData, double* inputData, int rows, int columns, int dataColumns, int* samples, int sampleCount){

    //If either pointer are not allocated, return
    if(outputParameters == nullptr || outputData == nullptr){
        return;
    }

    //For all sample rows
    for(int i = 0; i < sampleCount; i++){
        //For all columns in the row
        for(int j = 0; j < columns; j++){
            //If the column index is within the parameters column
            if(j < columns - dataColumns){
                //Copy the value to the parameters matrix
                outputParameters[colMajIndex(samples[i], j, columns-dataColumns)] = inputData[colMajIndex(samples[i], j, columns)];   
            }
            else{
                //Copy the value to the data matrix
                outputData[colMajIndex(samples[i], j, dataColumns)] = inputData[colMajIndex(samples[i], j, columns)];
            }
        }
    }
    return;
}

double kriging(std::string filename, int headerLines, int dataColumns, double theta, double variance, double nuggetEffect){
    //Begin variable definitions for data to be passed to GPU
    hipError_t cudaStatus = hipSuccess;
    int matrixMemoryAllocationSize = pow(dimension+1, 2);
    int vectorMemoryAllocationSize = dimension+1;
    double outputValue = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess){
        goto KrigingError;
    }

    //Create a dynamic allocation of memory for the identity matrix and populate with values
    double* identityMatrix = (double*) malloc(matrixMemoryAllocationSize * sizeof(double));
    
    //Create a dynamic allocation of memory for a temporary holding matrix
    double* tempMatrixOne = (double*) malloc(extendedMatrixMemoryAllocationSize * sizeof(double));
    double* tempMatrixTwo = (double*) malloc(extendedMatrixMemoryAllocationSize * sizeof(double));
    for (int i = 0; i < pow(dimension+1,2); i++) {
        tempMatrixOne[i] = 0;
        tempMatrixTwo[i] = 0;
        identityMatrix[i] = 0;
    }

KrigingError:
   
    free(identityMatrix);
    free(tempMatrixOne);
    free(tempMatrixTwo);
    
    return outputValue;
}

double metamodelSetup(int dimension, double theta, double variance, double a, double* designSite, double* testSite, double* designSiteValues) {
    
    //Begin variable definitions for data to be passed to GPU
    hipError_t cudaStatus = hipSuccess;
    int matrixMemoryAllocationSize = pow(dimension+1, 2);
    int extendedMatrixMemoryAllocationSize = pow(dimension+1, 2);
    int vectorMemoryAllocationSize = dimension+1;
    int extendedVectorMemoryAllocationSize = dimension + 1;
    double outputValue = 0;

    //Create a dynamic allocation of memory for the identity matrix and populate with values
    double* identityMatrix = (double*) malloc(matrixMemoryAllocationSize * sizeof(double));
    
    //Create a dynamic allocation of memory for a temporary holding matrix
    double* tempMatrixOne = (double*) malloc(extendedMatrixMemoryAllocationSize * sizeof(double));
    double* tempMatrixTwo = (double*) malloc(extendedMatrixMemoryAllocationSize * sizeof(double));
    for (int i = 0; i < pow(dimension+1,2); i++) {
        tempMatrixOne[i] = 0;
        tempMatrixTwo[i] = 0;
        identityMatrix[i] = 0;
    }

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess){
        goto SetupError;
    }

    //Create identity matrix on GPU, input is also the output
    cudaStatus = createIdentityMatrix(identityMatrix, dimension);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Calculate distance between design sites and values at design sites. tempMatrixOne will hold the output
    cudaStatus = calculateDistanceBetweenMatrices(tempMatrixOne, designSite, designSiteValues, dimension);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Calculate distance between test site and design site. tempMatrixTwo will hold the output
    cudaStatus = calculateDistanceBetweenMatrixVector(tempMatrixTwo, designSite, testSite, dimension);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Calculate the covariance between design sites and values at design sites. tempMatrixOne will hold the output
    cudaStatus = calculateGaussianCorrelation(tempMatrixOne, tempMatrixOne, variance, a, theta, dimension);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Calculate the covariance between test sites and design sites. tempMatrixTwo will hold the output
    cudaStatus = calculateGaussianCorrelation(tempMatrixTwo, tempMatrixTwo, variance, a, theta, dimension);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Clear the extraneous elements of the matrix after the valid vector elements to prevent later calculation errors.
    for (int i = dimension; i < pow(dimension + 1, 2); i++) {
        tempMatrixTwo[i] = 0;
    }

    //Extend the covariance matrix between the design site and design site values
    cudaStatus = extendMatrix(tempMatrixOne, tempMatrixOne, dimension);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Extend the covariance vector between test site and design sites. 
    tempMatrixTwo[dimension + 0*dimension] = 1; //Add 1 to the last row of the matrix, unclear if this is correct
    
    //Calculate inverse of extended covariance matrix
    cudaStatus = invertMatrix(identityMatrix, tempMatrixOne, dimension + 1);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Calculate extended weights vector, tempMatrixTwo will hold the result
    cudaStatus = calculateWeightVector(tempMatrixTwo, tempMatrixOne, tempMatrixTwo, dimension + 1);
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Calculate estimate value at test site, the ultimate output. 
    cudaStatus = multiplyMatrices(tempMatrixTwo, designSiteValues, dimension + 1); //Only consider elements within the dimension as the final value in the weight matrix is the lamda value (AKA not needed)
    if (cudaStatus != hipSuccess) {
        goto SetupError;
    }

    //Grab the first, and only, value of tempMatrixTwo as the ultimate output value
    outputValue = tempMatrixTwo[0];

    //Define error state
    //Need to manage/report error state so that the output value can be returned NOT an error status 
SetupError:
   if (cudaStatus != hipSuccess) {
        std::cout << "SETUP: Device failed" << std::endl;
        std::cout << "SETUP: CUDA Error Code -> " << hipGetErrorString(cudaStatus) << std::endl;
    }
    free(identityMatrix);
    free(tempMatrixOne);
    free(tempMatrixTwo);
    
    return outputValue;
}

//Use the outputMatrix is initially an input, but the matrix is then overwritten for the output to save memory. Unclear if this is entirely necessary to prevent overflowing memory.
hipError_t calculateGaussianCorrelation(double* outputMatrix, double* inMatrix, double variance, double a, double theta, int dimension) {
    
    //Allocate two pointers that will be used by the GPU for calculations.
    double* deviceOutMat = 0;
    double* deviceInMat = 0;

    //Create status variable for errors
    hipError_t cudaStatus = hipSuccess;

    //Create two integers that represent size to allocate on the GPU for matrices and vectors
    int matrixMemoryAllocationSize = pow(dimension, 2);
    int vectorMemoryAllocationSize = dimension;

    //Allocate memory on the GPU for the appropriate matrices
    cudaStatus = hipMalloc((void**)&deviceOutMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto CorrError;
    }

    cudaStatus = hipMalloc((void**)&deviceInMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto CorrError;
    }
    
    //Copy the CPU matrices to the GPU to allow for calculations
    cudaStatus = hipMemcpy(deviceInMat, inMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto CorrError;
    }

    cudaStatus = hipMemcpy(deviceOutMat, outputMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto CorrError;
    }
    
    //Call the GPU function with the appropriate number of blocks and threads to perform calculation on the GPU. Catch any error returned.
    calcGaussCorr <<< dimension, dimension >>> (deviceOutMat, deviceInMat, dimension, variance, a, theta);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto CorrError;
    }

    //Synchronize device to enable a copy of the result
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto CorrError;
    }

    //Copy data from the GPU to the CPU matrix.
    cudaStatus = hipMemcpy(outputMatrix, deviceOutMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto CorrError;
    }

CorrError:
    hipFree(deviceOutMat);
    hipFree(deviceInMat);

    return cudaStatus;
}

hipError_t calculateDistanceBetweenMatrices(double* outputMatrix, double* inMatrixOne, double* inMatrixTwo, int dimension) {
    double* deviceOutMat = 0;
    double* deviceInMatOne = 0;
    double* deviceInMatTwo = 0;
    int* deviceDimension = 0;
    hipError_t cudaStatus = hipSuccess;
    int matrixMemoryAllocationSize = pow(dimension, 2);

    //Begin allocation of memory on device
    cudaStatus = hipMalloc((void**)&deviceOutMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

    cudaStatus = hipMalloc((void**)&deviceInMatOne, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

    cudaStatus = hipMalloc((void**)&deviceInMatTwo, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

    //Begin copy of CPU local data to GPU
    cudaStatus = hipMemcpy(deviceInMatOne, inMatrixOne, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

    cudaStatus = hipMemcpy(deviceInMatTwo, inMatrixTwo, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

    //Perform calculation on the GPU and catch any error
    calcDistanceBetMats <<< dimension, dimension >>> (deviceOutMat, deviceInMatOne, deviceInMatTwo, dimension);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

    //Synchronize device to enable a copy of the result
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

    //Copy data from GPU address to CPU address
    cudaStatus = hipMemcpy(outputMatrix, deviceOutMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto MatDistError;
    }

MatDistError:
    hipFree(deviceOutMat);
    hipFree(deviceInMatOne);
    hipFree(deviceInMatTwo);

    return cudaStatus;
}

hipError_t calculateDistanceBetweenMatrixVector(double* outputMatrix, double* inMatrix, double* inVector, int dimension) {
    double* deviceOutMat = 0;
    double* deviceInMat = 0;
    double* deviceInVec = 0;
    hipError_t cudaStatus = hipSuccess;
    int matrixMemoryAllocationSize = pow(dimension, 2);

    //Begin allocation of memory on device
    cudaStatus = hipMalloc((void**)&deviceOutMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

    cudaStatus = hipMalloc((void**)&deviceInMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

    cudaStatus = hipMalloc((void**)&deviceInVec, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

    //Begin copy of CPU local data to GPU
    cudaStatus = hipMemcpy(deviceInMat, inMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

    cudaStatus = hipMemcpy(deviceInVec, inVector, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

    //Perform calculation on the GPU and catch any error
    calcDistanceBetMatVec <<< dimension, 1 >>> (deviceOutMat, deviceInMat, deviceInVec, dimension);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

    //Synchronize device to enable a copy of the result
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

    //Copy data from GPU address to CPU address
    cudaStatus = hipMemcpy(outputMatrix, deviceOutMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto VecDistError;
    }

VecDistError:
    hipFree(deviceOutMat);
    hipFree(deviceInMat);
    hipFree(deviceInVec);

    return cudaStatus;
}

hipError_t extendMatrix(double* outputMatrix, double* inputMatrix, int dimension) {
    double* deviceOutMat = 0;
    double* deviceInMat = 0;
    hipError_t cudaStatus = hipSuccess;
    int matrixMemoryAllocationSize = pow(dimension+1, 2);

    //Swap matrix locations to prepare for "extension" of the matrix.  
    for (int i = dimension; i >= 0; i--) {
        for (int j = dimension; j >= 0; j--) {
            //printf("OLD [%d], NEW [%d]\n", (i + j * dimension), (i + j * (dimension + 1)));
            inputMatrix[i + j * (dimension + 1)] = inputMatrix[i + j * dimension];
        }
    }

    //Begin allocation of memory on GPU device
    cudaStatus = hipMalloc((void**)&deviceOutMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto ExtendError;
    }

    cudaStatus = hipMalloc((void**)&deviceInMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto ExtendError;
    }

    //Begin copy of CPU local data to GPU
    cudaStatus = hipMemcpy(deviceInMat, inputMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto ExtendError;
    }

    //Perform calculation on the GPU and catch any error
    extendMat <<<dimension+ 1 , dimension+ 1>>> (deviceOutMat, deviceInMat, dimension+1); 
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto ExtendError;
    }

    //Synchronize device to enable a copy of the result
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto ExtendError;
    }

    //Copy data from GPU address to CPU address
    cudaStatus = hipMemcpy(outputMatrix, deviceOutMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto ExtendError;
    }

ExtendError:
    hipFree(deviceOutMat);
    hipFree(deviceInMat);

    return cudaStatus;
}

hipError_t invertMatrix(double* outputMatrix, double* inputMatrix, int dimension) {
    
    double* deviceInMat = 0;
    double* deviceIdenMat = 0;
    double* deviceBuffer = 0;
    bool* deviceFlag = 0;
    bool* flag = new bool;
    *flag = true;

    int matrixMemoryAllocationSize = pow(dimension, 2);
    hipError_t cudaStatus = hipSuccess;

    cudaStatus = hipMalloc((void**)&deviceBuffer, 2*dimension * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMalloc((void**)&deviceFlag, sizeof(bool));
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMalloc((void**)&deviceInMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMalloc((void**)&deviceIdenMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMemcpy(deviceFlag, flag, sizeof(bool), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMemcpy(deviceInMat, inputMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMemcpy(deviceIdenMat, outputMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    for (int i = 0; i < dimension; i++) {
        resetBuffers << <dimension, dimension >> > (deviceBuffer, deviceFlag, dimension);
        normalizeRows << <dimension, dimension >> > (deviceIdenMat, deviceInMat, deviceBuffer, deviceFlag, dimension, i);
        pivotDown << <dimension, dimension >> > (deviceIdenMat, deviceInMat, dimension, i);
        
    }

    for (int i = dimension - 1; i > 0; i--) {
        resetBuffers << <dimension, dimension >> > (deviceBuffer, deviceFlag, dimension);
        pivotUp << <dimension, dimension >> > (deviceIdenMat, deviceInMat, deviceBuffer, deviceFlag, dimension, i);
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    //Synchronize device to enable a copy of the result
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMemcpy(outputMatrix, deviceIdenMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

    cudaStatus = hipMemcpy(inputMatrix, deviceInMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto InvertError;
    }

InvertError:
    hipFree(deviceIdenMat);
    hipFree(deviceInMat);
    hipFree(deviceBuffer);
    hipFree(deviceFlag);
    free(flag);

    return cudaStatus;
}

hipError_t createIdentityMatrix(double* matrix, int dimension) {
    double* deviceMat = 0;
    hipError_t cudaStatus = hipSuccess;
    int matrixMemoryAllocationSize = pow(dimension, 2);

    cudaStatus = hipMalloc((void**)&deviceMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto IdentityError;
    }

    cudaStatus = hipMemcpy(deviceMat, matrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto IdentityError;
    }

    createIdentMat <<<dimension, dimension >> > (deviceMat, dimension);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto IdentityError;
    }

    cudaStatus = hipMemcpy(matrix, deviceMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto IdentityError;
    }


IdentityError:
    hipFree(deviceMat);

    return cudaStatus;
}

hipError_t calculateWeightVector(double* outputVectorMatrix, double* invertedCovarianceMatrix, double* covarianceVectorMatrix, int dimension) {
    double* deviceOutMat = 0;
    double* deviceInvCovMat = 0;
    double* deviceCovVecMat = 0;
    hipError_t cudaStatus = hipSuccess;
    int matrixMemoryAllocationSize = pow(dimension, 2);
    int vectorMemoryAllocationSize = dimension;

    //Begin GPU memory allocation
    cudaStatus = hipMalloc((void**)&deviceOutMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }

    cudaStatus = hipMalloc((void**)&deviceInvCovMat, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }

    cudaStatus = hipMalloc((void**)&deviceCovVecMat, vectorMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }

    //Copy data from local CPU to GPU
    cudaStatus = hipMemcpy(deviceInvCovMat, invertedCovarianceMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }

    cudaStatus = hipMemcpy(deviceCovVecMat, covarianceVectorMatrix, vectorMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }

    multiplyMatrix << <dimension, dimension >> > (deviceOutMat, deviceInvCovMat, deviceCovVecMat, dimension);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }

    cudaStatus = hipMemcpy(outputVectorMatrix, deviceOutMat, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto WeightError;
    }


WeightError:
    hipFree(deviceOutMat);
    hipFree(deviceInvCovMat);
    hipFree(deviceCovVecMat);

    return cudaStatus;
}

hipError_t multiplyMatrices(double* output, double* inputMatrix, int dimension) {
    double* deviceInputOne = 0;
    double* deviceInputTwo = 0;
    double* deviceOutput = 0;
    int matrixMemoryAllocationSize = pow(dimension, 2);
    hipError_t cudaStatus = hipSuccess;

    cudaStatus = hipMalloc((void**)&deviceInputOne, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto MultiplyError;
    }

    cudaStatus = hipMalloc((void**)&deviceInputTwo, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto MultiplyError;
    }

    cudaStatus = hipMalloc((void**)&deviceOutput, matrixMemoryAllocationSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        goto MultiplyError;
    }

    cudaStatus = hipMemcpy(deviceInputOne, output, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto MultiplyError;
    }

    cudaStatus = hipMemcpy(deviceInputTwo, inputMatrix, matrixMemoryAllocationSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        goto MultiplyError;
    }

    multiplyMatrix << <dimension, dimension >> > (deviceOutput, deviceInputOne, deviceInputTwo, dimension);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        goto MultiplyError;
    }

    cudaStatus = hipMemcpy(output, deviceOutput, matrixMemoryAllocationSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto MultiplyError;
    }

MultiplyError:
    hipFree(deviceInputOne);
    hipFree(deviceInputTwo);
    hipFree(deviceOutput);

    return cudaStatus;
}

void printMatrix(double inArray[], int dimension) {
    int index = 0;
    std::cout << "{";
    for (int i = 0; i < dimension; i++) {
        for (int j = 0; j < dimension; j++) {
            index = colMajIndex(i,j,dimension);
            //printf("printMatrix[%d] = %f\n", index, inArray[index]);

            if (j + 1 < dimension) {
                printf("%0.5f,", inArray[index]);
            }
            else {
                printf("%0.5f", inArray[index]);
            }

        }

        if (i + 1 < dimension) {
            printf(";\n");
        }
        else {
            printf("}\n");
        }

    }
    return;
}

void printMatrix(double inArray[], int numRows, int numColumns) {
    
    int index = 0;
    for (int i = 0; i < numRows; i++) {
        for (int j = 0; j < numColumns; j++) {
            index = colMajIndex(i,j,numRows);
            //printf("printMatrix[%d] = %f\n", index, inArray[index]);

            if (j + 1 < numColumns) {
                printf("%0.5f,", inArray[index]);
            }
            else {
                printf("%0.5f", inArray[index]);
            }

        }

        if (i + 1 < numRows) {
            printf(";\n");
        }
        else {
            printf("}\n");
        }

    }
    return;

}

//Begin CUDA Function Implementations
__global__ void calcDistance(double* outMat, double* inMat, int inRows, int inCols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    //Generate thread index based on input columns
    int index = i + j * inCols;
    //Calculate specific i and j parameters based on input rows/cols
    i = index % inCols;
    j = index / inCols;
    //For all columns in the input matrix, 
    for(int k = 0; k < inRows){
    	//Add the difference of the values to the output matrix
        outMat[index] += std::pow(inMat[k + i*inCols] - inMat[k + j*inCols], 2);
    }
    return;
}

__global__ void calcGaussCorr(double* outMat, double* inMat, int dimension, double variance, double a, double theta) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * dimension;
    //Artifact of the hipComplex purge. Leaving here as it might be necessary later and will save a little time.
    double negOne = -1;
    outMat[index] = (variance - a) * std::exp(negOne * theta * inMat[index]);
    
    return;
}

__global__ void resetBuffers(double* vals, bool* flag, int dimension) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * dimension;

    //Reset the flag to false if the zeroth index. This prevents race conditions between threads as there is only one value to write. 
    if (index == 0) {
        *flag = false;
    }

    //If the index is below double the dimensions, set the value equal to zero. Not strictly necessary, but prevents data manipulation by extra threads in the case of thread optimization.
    if (index < 2*dimension) {
        vals[index] = 0;
    }
    return;
}

__global__ void normalizeRows(double* idenMat, double* inMat, double* firstVals, bool* flag, int dimension, int targetCol) {
    /*BEGIN DOC
    Function:       normalizeRows(idenMat, inMat, dimension, flag, targetCol, firstVals)
    Description:    Perform the necessary normalization for the pivot down step of the Gauss-Jordan Elimination algorithm.
    Inputs:         identMat (double*) - A pointer to an array of double values that represent an identity matrix. Stored in row major format
                    inMat (double*) - A pointer to an array of double values that represent a matrix of values to invert. Stored in row major format.
                    dimension (int) - The dimension, n, of the square matrices.
                    flag (bool*) - A pointer to a boolean that functions as the flag to signal all threads to complete operations.
                    targetCol (int) - The target column index from which the pivot downwards operation occurs.
    Outputs:        None
    Notes:          Each row will be normalized by the value in the target column. The target column value is also used as a row index modifier to prevent the further normalization of values on the diagonal that have a lower index that the leading value of the targeted column.
    END DOC*/

    //Get the index value for the particular thread by 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * dimension;
    
    //If the thread index is that of the first row and targeted column
    if (index == targetCol * (dimension + 1)) {
        //Get all subsequent element in the column of index
        for (int k = index % dimension; k < dimension; k++) {
            firstVals[k] = inMat[index - targetCol + k];
        }
        *flag = true;
    }

    //Use the flag as a lock to prevent divide-by-zero errors
    while (*flag == false) {
        //Simply wait for the array of first values to be set. 
    }

    //Grab the normalizing value for the particular thread by dividing the index by dimension in integer division. Decimal values should not be present.
    double normVal = firstVals[index % dimension];
    //If normalizing value is not zero and the index falls within the desired submatrix
    if (normVal != 0 && index >= targetCol % dimension) { //normVal is a double, might not ever equal 0. Might need to figure out a better way to exclude values close to zero. Establish precision limits.
        //Divide the identity and input matrix position by the normalizing value
        idenMat[index] /= normVal;
        inMat[index] /= normVal;
    }

    return;
}
    
__global__ void pivotDown(double* idenMat, double* inMat, int dimension, int targetRow) {
    /*BEGIN DOC
    Function:       pivotDown(idenMat, inMat, dimension, flag, targetRow)
    Description:    Perform the pivot down step of the Gauss-Jordan Elimination algorithm. 
    Inputs:         identMat (double*) - A pointer to an array of double values that represent an identity matrix. Stored in row major format
                    inMat (double*) - A pointer to an array of double values that represent a matrix of values to invert. Stored in row major format.
                    dimension (int) - The dimension, n, of the square matrices. 
                    targetRow (int) - The target row index from which the pivot downwards operation occurs.
    Outputs:        None
    Notes:          Every row below the target row will have their column values subtracted by the pivot row's respective column value. The pivot row remains the same. 
    END DOC*/

    //Get the index value for the particular thread by multiplying the thread index values with the dimension in a row major format
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * dimension;

    //If the current index falls above the targeted row and the leading element is not zero, subtract the target row column value from the index's value
    if ( (index % dimension > targetRow) && (inMat[index % dimension + targetRow*dimension] != 0)) {
        //double temp = inMat[index - (index % dimension - targetRow)];
        int temp = index - (index % dimension - targetRow);
        idenMat[index] -= idenMat[temp];
        inMat[index] -= inMat[temp];
    }
    return;
}

__global__ void pivotUp(double* idenMat, double* inMat, double* lastVals, bool* flag, int dimension, int targetRow) {
    /*BEGIN DOC
    Function:       pivotUp(idenMat, inMat, lastVals, flag, dimension, targetRow)
    Description:    Perform the pivot up step of the Gauss-Jordan Elimination algorithm.
    Inputs:         identMat (double*) - A pointer to an array of double values that represent an identity matrix. Stored in row major format
                    inMat (double*) - A pointer to an array of double values that represent a matrix of values to invert. Stored in row major format.
                    dimension (int) - The dimension, n, of the square matrices.
                    flag (bool*) - A pointer to a boolean that functions as the flag to signal all threads to complete operations.
                    targetRow (int) - The target row index from which the pivot upwards operation occurs. 
                    lastVals (double*) - A pointer to an array that will be used to store values for operations.
    Outputs:        None
    Notes:          Every row above the target row will have their column values subtracted by the pivot row's respective column value multiplied by the column value indicated by the targetRow value for that row. As the matrices are square, the target row is also the target column. The pivot row remains the same.
    END DOC*/

    //Get the index value for the particular thread by 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * dimension;

    //From the leading pivot column, find non-zero values from this row to the zeroth row
    if (index == targetRow * (dimension + 1)) {
        for (int k = index % dimension; k >= 0; k--) {
            lastVals[(index - k)%dimension] = inMat[index - k];
        }
        *flag = true;
    }

    while (*flag == false) {
        //Wait for the lastVals array to be populated
    }

    //If the index falls in a row below the target, perform subtraction operations
    if (index % dimension < targetRow ) {
        //Multiply the index value by the trailing row value and subtract value from current index in both matrices
        idenMat[index] -= lastVals[index % dimension] * idenMat[index + targetRow - (index % dimension)];
        inMat[index] -= lastVals[index % dimension] * inMat[index + targetRow - (index % dimension)];
    }
    return;
}

//This might yield race condition errors
__global__ void multiplyMatrix(double* output, double* firInput, double* secInput, int dimension) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    //Generate index for the output based on thread and block parameters
    int index = i + j * dimension;
    
    //Calculate target row and column values from the index to allow for dot product to occur
    int row = index % dimension;
    int col = index / dimension;

    //Multiply each element pair and then sum them together
    for (int k = 0; k < dimension; k++) {
        output[index] += firInput[row + k * dimension] * secInput[k +  col * dimension];
    }

    return;
}

__global__ void extendMat(double* outMat, double* inMat, int dimension) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * dimension;
    
    //If the current index is on the bottom-most row of the extended matrix
    if ( ((index + 1) % dimension) == 0) {
        
        //If the sqrt(index+1) is equal to the dimension, it is the bottom right corner of the extended matrix
        if ((int)sqrt((float)(index + 1)) == dimension) {
            outMat[index] = 0.0;
        }
        //Otherwise, it is the bottom row of the extended matrix
        else {
            outMat[index] = 1.0;
        }
    }
    //If the current index is on the right-most column of the extended matrix. The previous statement should get the bottom right coordinate which is set to zero.
    else if ( index >= (dimension)*(dimension-1)) {
        
        outMat[index] = 1.0;
    }
    //Otherwise, the index is within the original matrix
    else {
        
        outMat[index] = inMat[index];
    }
    return;
}

__global__ void normalizeMatrix(double* outMat, double* inMat, double normalizingValue, int dimension) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    outMat[i + j * dimension] = inMat[i + j * dimension] / normalizingValue;

    return;
}

__global__ void createIdentMat(double* matrix, int dimension) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * dimension;

    if (index % (dimension + 1) == 0) {
        matrix[index] = 1;
    }
    else {
        matrix[index] = 0;
    }

    return;
}