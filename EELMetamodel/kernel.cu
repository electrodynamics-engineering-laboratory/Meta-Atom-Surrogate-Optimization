#include "hip/hip_runtime.h"
/*
File: kernel.cu
Date: 2-24-2020
Author: Joe Haun
Purpose: This file is for the purposes of writing Metamodel test code. 
*/

#include "hip/hip_runtime.h"
#include ""
#include "EELSurrogate.cu"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <complex>


int main()
{
    const int arraySize = 4;
    double dummyDesignSite[arraySize] = { 1, 2, 3, 4 };
    double dummyTestSite[arraySize] = { 10, 11, 0, 0};
    double dummyDesignSiteValues[arraySize] = { 6, 7, 8, 9 };
    double result = -1;

    double theta = 1;
    double aVal = 1;
    double variance = 0;
    hipError_t cudaStatus = hipSuccess;

    
    //Perform metamodel Kriging operations
    result = metamodelSetup(sqrt(arraySize), theta, variance, aVal, dummyDesignSite, dummyTestSite, dummyDesignSiteValues);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel failed!");
        return 1;
    }

    std::cout << "MAIN: Matrices after calculations." << std::endl;
    //Print first input matrix
    printf("MAIN: designSite\n");
    printMatrix(dummyDesignSite, sqrt(arraySize));
    //Print second input matrix
    printf("MAIN: testSite\n");
    printMatrix(dummyTestSite, sqrt(arraySize));
    //Print output matrix
    printf("MAIN: designSiteValues\n");
    printMatrix(dummyDesignSiteValues, sqrt(arraySize));

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
   
    return 0;
}



